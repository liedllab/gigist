#include "GistCudaSetup.cuh"
#include "GistCudaCalc.cuh"
#include "EntropyCalculator.cuh"
#include <iostream>


/**
 * Allocate memory on the GPU.
 * @parameter array: The pointer to the array, which will be allocated on the GPU.
 * @parameter size: An integer giving the size of the array, which will be allocated.
 * @throws: CudaException if a problem occurs.
 */
__host__
void allocateCuda(void **array, int size) {
  // Check if the array is actually free, if not, it will be freed 
  // (fun fact: checking is not necessary, one could also simply free the memory).
  if ((*array) != NULL) {
    hipFree(*array);
  }
  // If something goes wrong, throw exception
  if (hipMalloc(array, size) != hipSuccess) {
    throw CudaException();
  }
}

/**
 * Copy memory from the CPU to the GPU.
 * @parameter array: The array from which the values shall be copied.
 * @parameter array_c: The array on the device, to which the values shall be copied.
 * @parameter size: The size of the stuff which will be copied.
 * @throws: CudaException if something goes wrong.
 */
__host__
void copyMemoryToDevice(void *array, void *array_c, int size) {
  // If something goes wrong, throw exception
  // In this case only copying can go wrong.
  if (hipMemcpy(array_c, array, size, hipMemcpyHostToDevice) != hipSuccess) {
    throw CudaException();
  }
}

/**
 * A simple helper function that copies a lot of stuff to the GPU (as structs).
 * @parameter charge: An array holding the charges for the different atoms.
 * @parameter atomtype: An array holding the integers for the atom types of the different atoms.
 * @parameter solvent: An array of boolean values, holding the information whether a certain atom is solvent or solute.
 * @parameter atomNumber: The total number of atoms.
 * @parameter atomProps_c: A pointer to an array on the GPU, which will hold the atom properties.
 * @parameter ljA: An array holding the lennard-jones parameter A for each atom type pair.
 * @parameter ljB: An array holding the lennard-jones parameter B for each atom type pair.
 * @parameter length: The length of the two aforementioned arrays (ljA & ljB).
 * @parameter lJparams_c: A pointer to an array on the GPU, which will hold the lj parameters.
 * @throws: CudaException if something bad happens.
 */
__host__
void copyMemoryToDeviceStruct(float *charge, int *atomtype, bool *solvent, int *molecule, int atomNumber, void **atomProps_c,
                              float *ljA, float *ljB, int length, void **lJparams_c) {
  // Check if the two arrays are free. Again, this could be removed (but will stay!)
  if ((*atomProps_c) != NULL) {
    hipFree(*atomProps_c);
  }
  if ((*lJparams_c) != NULL) {
    hipFree(*lJparams_c);
  }
  // Allocate the necessary memory on the GPU.
  if (hipMalloc(atomProps_c, atomNumber * sizeof(AtomProperties)) != hipSuccess) {
    throw CudaException();
  }
  if (hipMalloc(lJparams_c, length * sizeof(ParamsLJ)) != hipSuccess) {
    throw CudaException();
  }

  // Create an array for the lennard-jones parameters.
  ParamsLJ *ljp = (ParamsLJ *) malloc (length * sizeof(ParamsLJ));
  // Add the lennard-jones parameters to the array.
  for (int i = 0; i < length; ++i) {
    ljp[i] = ParamsLJ(ljA[i], ljB[i]);
  }

  // Create an array for the atom properties.
  AtomProperties *array = (AtomProperties *)malloc(atomNumber * sizeof(AtomProperties));
  // Add the properties into the array.
  for (int i = 0; i < atomNumber; ++i) {
    array[i] = AtomProperties(charge[i], atomtype[i], solvent[i], molecule[i]);
  }
  // Copy the memory from the host to the device.
  if (hipMemcpy((*atomProps_c), array, atomNumber * sizeof(AtomProperties), hipMemcpyHostToDevice) != hipSuccess) {
    throw CudaException();
  }
  if (hipMemcpy((*lJparams_c), ljp, length * sizeof(ParamsLJ), hipMemcpyHostToDevice) != hipSuccess) {
    throw CudaException();
  }

  // Free the two arrays (so that no memory leak occurs).
  free(ljp);
  free(array);
}

/**
 * Free an array.
 * @parameter array: The array you want to free.
 */
__host__
void freeCuda(void *array) {
  hipFree(array);
}


// This is coded C-like, but uses exceptions.
/**
 * This starts the cuda kernel, thus it is actually a quite long function.
 */
__host__
std::vector<std::vector<float> > doActionCudaEnergy(const double *coords, int *NBindex_c, int ntypes, void *parameter, void *molecule_c,
                            int boxinfo, float *recip_o_box, float *ucell, int maxAtoms, float *min_c, float *max_c, int headAtomType, 
                            float neighbourCut2, int *result_o, int *result_n, float *result_w_c, float *result_s_c,
                            int *result_O_c, int *result_N_c, bool doorder) {
  Test *coords_c   = NULL;
  float *recip_b_c  = NULL;
  float *ucell_c    = NULL;
  
  

  float *result_A = (float *) calloc(maxAtoms, sizeof(float));
  float *result_s = (float *) calloc(maxAtoms, sizeof(float));
  // TODO: Fix this, test is actually a quite bad name here!
  Test *coord_array = (Test *) calloc(maxAtoms, sizeof(Test));
  
  // Casting
  AtomProperties *sender = (AtomProperties *) molecule_c;
  ParamsLJ *lennardJonesParams = (ParamsLJ *) parameter;
  
  // Create Boxinfo and Unit cell. This is actually very important for the speed (otherwise
  // there would be LOTS of access to non-local variables).
  BoxInfo boxinf;
  if (boxinfo != 0) {
    boxinf = BoxInfo(recip_o_box, boxinfo);
  }
  UnitCell ucellN;
  if (boxinfo == 2) {
    ucellN = UnitCell(ucell);
  }
  
  // Add the coordinates to the array.
  // TODO: Fix Test here also!
  for (int i = 0; i < maxAtoms; ++i) {
    coord_array[i] = Test(&coords[i * 3]);
  }

  // vectors that will return the necessary information.
  std::vector<std::vector<float> > result;
  std::vector<float> result_esw;
  std::vector<float> result_eww;

  // Allocate space on the GPU
  if (hipMalloc(&coords_c, maxAtoms * sizeof(Test)) != hipSuccess) {
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }


  // Copy the data to the GPU
  if (hipMemcpy(coords_c, coord_array, maxAtoms * sizeof(Test), hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }
  if (hipMemcpy(result_w_c, result_A, maxAtoms * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }
  if (hipMemcpy(result_s_c, result_s, maxAtoms * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }

  // If the doorder calculation is used, it needs to calculate everything differently, so the slow version is used
  // (this is about 10% slower).
  if (doorder) {
    cudaCalcEnergySlow<<< (maxAtoms + SLOW_BLOCKSIZE) / SLOW_BLOCKSIZE, SLOW_BLOCKSIZE >>> (coords_c, NBindex_c, ntypes, lennardJonesParams, sender,
                                                                                            boxinf, ucellN, maxAtoms, result_w_c, result_s_c, min_c, max_c,
                                                                                            headAtomType, neighbourCut2, result_O_c, result_N_c);
  } else {
    // Uses a 2D array, which is nice for memory access.
    dim3 threadsPerBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 numBlocks((maxAtoms + threadsPerBlock.x) / threadsPerBlock.x, (maxAtoms + threadsPerBlock.y) / threadsPerBlock.y);
    // The actual call of the device function
    cudaCalcEnergy<<<numBlocks, threadsPerBlock>>> (coords_c, NBindex_c, ntypes, lennardJonesParams, sender,
                                                                      boxinf, ucellN, maxAtoms, result_w_c, result_s_c, min_c, max_c,
                                                                      headAtomType, neighbourCut2, result_O_c, result_N_c);
    // Check if there was an error.
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
      printf("returned %s\n", hipGetErrorString(hipError_t));
    }
  }
  // Return the results of the calculation to the main memory
  if (hipMemcpy(result_A, result_w_c, maxAtoms * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }  
  

  if (hipMemcpy(result_s, result_s_c, maxAtoms * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }


  
  if (hipMemcpy(result_o, result_O_c, maxAtoms * 4 * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }
  
  if (hipMemcpy(result_n, result_N_c, maxAtoms * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }

  for (int i = 0; i < maxAtoms; ++i) {
    result_eww.push_back(result_A[i]);
    result_esw.push_back(result_s[i]);
  }

  result.push_back(result_eww);
  result.push_back(result_esw);

  // Free everything used in here.
  hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
  free(result_A); free(result_s); free(coord_array);
  
  return result;
}

#ifdef DEBUG_GIST_CUDA
// Not necessary
__host__
std::vector<Quaternion<float> > shoveQuaternionsTest(std::vector<Quaternion<float> > quats) {
  QuaternionG<float> *quats_c = NULL;
  float *ret_c = NULL;
  std::vector<Quaternion<float> > ret;
  float *ret_f = new float[quats.size() * 4];
  QuaternionG<float> *quats_f = new QuaternionG<float>[quats.size()];
  for (int i = 0; i < quats.size(); ++i) {
    quats_f[i] = quats.at(i);
  }
  if (hipMalloc(&quats_c, quats.size() * sizeof(QuaternionG<float>)) != hipSuccess) {
    delete quats_f; delete ret_f;
    throw CudaException();
  }
  if (hipMalloc(&ret_c, quats.size() * 4 * sizeof(float)) != hipSuccess) {
    hipFree(quats_c);
    delete quats_f; delete ret_f;
    throw CudaException();
  }

  if (hipMemcpy(quats_c, quats_f, quats.size() * sizeof(QuaternionG<float>), hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(quats_c); hipFree(ret_c);
    delete quats_f; delete ret_f;
    throw CudaException();
  }

  shoveQuaternions<<< (quats.size() + BLOCKSIZE) / BLOCKSIZE, BLOCKSIZE >>> (quats_c, quats.size(), ret_c);

  if (hipMemcpy(ret_f, ret_c, quats.size() * 4 * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(quats_c); hipFree(ret_c);
    delete quats_f; delete ret_f;
    throw CudaException();
  }

  for (int i = 0; i < quats.size(); ++i) {
    ret.push_back(Quaternion<float>(ret_f[i * 4], ret_f[i * 4 + 1], ret_f[i * 4 + 2], ret_f[i * 4 + 3]));
  }

  hipFree(quats_c); hipFree(ret_c);
  delete quats_f; delete ret_f;

  

  return ret;
}
#endif

/**
 * Calculates the entropy on the GPU (this is not really necessary and does not lead to a significant speed up).
 * @parameter coords: The coordinates of the different water molecules.
 * @parameter x: The number of grid voxels in the x direction.
 * @parameter y: The number of grid voxels in the y direction.
 * @parameter z: The number of grid voxels in the z direction.
 * @parameter quats: A vector object holding all the quaternions.
 * @parameter temp: The temperature.
 * @parameter rho0: The reference density.
 * @parameter nFrames: The total number of frames.
 * @return: A vector holding the values for dTStrans, dTSorient and dTSsix.
 * @throws: A CudaException on error.
 */
std::vector<std::vector<float> > doActionCudaEntropy(std::vector<std::vector<Vec3> > coords, int x, int y, int z, std::vector<std::vector<Quaternion<float> > > quats, float temp, float rho0, int nFrames) {
  
  // For the CPU
  // Input (from previous calculations)
  std::vector<QuaternionG<float> > quatsF;
  std::vector<float> coordsF;
  std::vector<int> cumSumAtoms;
  // Results
  float *resultTStrans  = new float[quats.size()];
  float *resultTSorient = new float[quats.size()];
  float *resultTSsix    = new float[quats.size()];

  // For the GPU
  // Input (from previous calculations)
  Dimensions dims            = Dimensions(x, y, z);
  float *coordsG             = NULL;
  QuaternionG<float> *quatsG = NULL;
  int *cumSumAtomsG          = NULL;
  // Results
  float *resultTStransG       = NULL;
  float *resultTSorientG      = NULL;
  float *resultTSsixG         = NULL;
  
  int sum = 0;
  for (int i = 0 ; i < quats.size(); ++i) {
    sum += quats.at(i).size();
    cumSumAtoms.push_back(sum);
    for (int j = 0; j < quats.at(i).size(); ++j) {
      // quatsF always has the size of the number of the current molecule. 
      coordsF.push_back((float) (coords.at(i).at(j)[0]));
      coordsF.push_back((float) (coords.at(i).at(j)[1]));
      coordsF.push_back((float) (coords.at(i).at(j)[2]));
      quatsF.push_back(quats.at(i).at(j));
    }
  }



  hipError_t err1 = hipMalloc(&quatsG, quatsF.size() * sizeof(QuaternionG<float>));
  hipError_t err2 = hipMalloc(&coordsG, coordsF.size() * sizeof(float));
  hipError_t err3 = hipMalloc(&cumSumAtomsG, cumSumAtoms.size() * sizeof(int));
  hipError_t err4 = hipMalloc(&resultTStransG, quats.size() * sizeof(float));
  hipError_t err5 = hipMalloc(&resultTSorientG, quats.size() * sizeof(float));
  hipError_t err6 = hipMalloc(&resultTSsixG, quats.size() * sizeof(float));
  // Error Check
  if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess ||
      err4 != hipSuccess || err5 != hipSuccess || err6 != hipSuccess) {
    hipFree(quatsG);
    hipFree(coordsG);
    hipFree(cumSumAtomsG);
    hipFree(resultTStransG);
    hipFree(resultTSorientG);
    hipFree(resultTSsixG);
    delete[] resultTStrans;
    delete[] resultTSorient;
    delete[] resultTSsix;
    throw CudaException();
  }


  err1 = hipMemcpy(quatsG, &(quatsF[0]), quatsF.size() * sizeof(QuaternionG<float>), hipMemcpyHostToDevice);
  err2 = hipMemcpy(coordsG, &(coordsF[0]), coordsF.size() * sizeof(float), hipMemcpyHostToDevice);
  err3 = hipMemcpy(cumSumAtomsG, &(cumSumAtoms[0]), cumSumAtoms.size() * sizeof(int), hipMemcpyHostToDevice);

  
  // Error Check
  if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
    hipFree(quatsG);
    hipFree(coordsG);
    hipFree(cumSumAtomsG);
    hipFree(resultTStransG);
    hipFree(resultTSorientG);
    hipFree(resultTSsixG);
    delete[] resultTStrans;
    delete[] resultTSorient;
    delete[] resultTSsix;
    throw CudaException();
  }

  EntropyCalculator entCalc = EntropyCalculator(quatsG, coordsG, dims, cumSumAtomsG, temp, rho0, nFrames);
  calculateEntropy<<<(quats.size() + SLOW_BLOCKSIZE) / SLOW_BLOCKSIZE, SLOW_BLOCKSIZE>>>(entCalc, resultTStransG, resultTSorientG, resultTSsixG);
  hipError_t err7 = hipGetLastError();

  // Error Check
  if (err7 != hipSuccess) {
    hipFree(quatsG);
    hipFree(coordsG);
    hipFree(cumSumAtomsG);
    hipFree(resultTStransG);
    hipFree(resultTSorientG);
    hipFree(resultTSsixG);
    delete[] resultTStrans;
    delete[] resultTSorient;
    delete[] resultTSsix;
    throw CudaException();
  }

  // Copy back, use same errors as above for understandability.
  err4 = hipMemcpy(resultTStrans, resultTStransG, quats.size() * sizeof(float), hipMemcpyDeviceToHost);
  err5 = hipMemcpy(resultTSorient, resultTSorientG, quats.size() * sizeof(float), hipMemcpyDeviceToHost);
  err6 = hipMemcpy(resultTSsix, resultTSsixG, quats.size() * sizeof(float), hipMemcpyDeviceToHost);

  // Don't need that anymore.
  hipFree(quatsG);
  hipFree(coordsG);
  hipFree(cumSumAtomsG);
  hipFree(resultTStransG);
  hipFree(resultTSorientG);
  hipFree(resultTSsixG);

  // Error Check
  if (err4 != hipSuccess || err5 != hipSuccess || err6 != hipSuccess) {
    delete[] resultTStrans;
    delete[] resultTSorient;
    delete[] resultTSsix;
    throw CudaException();
  }

  std::vector<float> trans;
  std::vector<float> orient;
  std::vector<float> six;

  for (int i = 0; i < quats.size(); ++i) {
    trans.push_back(resultTStrans[i]);
    orient.push_back(resultTSorient[i]);
    six.push_back(resultTSsix[i]);
  }

  std::vector<std::vector<float> > ret;
  ret.push_back(trans);
  ret.push_back(orient);
  ret.push_back(six);

  delete[] resultTStrans;
  delete[] resultTSorient;
  delete[] resultTSsix;

  return ret;
}
